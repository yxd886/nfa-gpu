#include "hip/hip_runtime.h"
#include "d_network_function_base.cuh"
#include "d_network_function_derived.cuh"
#include "d_nf_item.h"
#include "../d_firewall/d_firewall.cuh"
#include "../d_flowmonitor/d_flow_monitor.cuh"
#include "../d_httpparser/d_http_parser.cuh"
#include "../d_pktcounter/d_pkt_counter.cuh"
#include "Pkt.h"
#include "d_nf_processor.cuh"


__device__ void Init_nfs(struct d_flow_actor_nfs* nfs,struct d_rule* firewall_rules){

    uint8_t i=3;

	nfs->nf[1]=new d_network_function_derived<d_pkt_counter, d_pkt_counter_fs>(1);
	nfs->nf[2]=new d_network_function_derived<d_flow_monitor, d_flow_monitor_fs>(2);
	nfs->nf[3]=new d_network_function_derived<d_firewall, d_firewall_fs>(3);
	nfs->nf[4]=new d_network_function_derived<d_http_parser, d_http_parser_fs>(4);

}


__device__ void Release_nfs(struct d_flow_actor_nfs* nfs){


	delete nfs->nf[1];
	delete nfs->nf[2];
	delete nfs->nf[3];
	delete nfs->nf[4];

}


__device__ uint8_t compute_network_function(uint64_t s, int pos){
  return static_cast<uint8_t>((s>>(8*pos))&0x00000000000000FF);
}

__device__ int compute_service_chain_length(uint64_t s){
  int length = 0;
  bool encounter_zero = false;

#pragma unroll
  for(int i=0; i<8; i++){
    uint8_t nf =
        static_cast<uint8_t>((s>>(8*i))&0x00000000000000FF);
    if(nf>0){
      length+=1;
      if(encounter_zero){
        return -1;
      }
    }
    else{
      encounter_zero = true;
    }
  }
  return length;
}


__global__ void
Runtask(Pkt* pkts, Fs* fs, uint64_t service_chain,int packet_num,int* flow_size,struct d_rule* firewall_rules)
{

	//__shared__ Pkt share_pkts[200*32*10];
	//__shared__ Fs share_fs[200*32];
	struct d_flow_actor_nfs  nfs;
	Init_nfs(&nfs,firewall_rules);
	int chain_len=compute_service_chain_length(service_chain);
	int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < packet_num)
    {

    	//int begin=flow_pos[i];
    	int num=flow_size[i];
    //	for(int j=0;j<num;j++){
    //		memcpy(&share_pkts[i+j*32],&pkts[i+j*32],sizeof(Pkt));

    //	}
    //	memcpy(&share_fs[i],&fs[i],sizeof(Fs));
    //	__Syncthreads();


    	//int j=i;
    	for(int j=0;j<num;j++){
    		//pkts[j].full=0;
    		for(int k=0; k<chain_len; k++){
    			int nf_id=compute_network_function(service_chain,k);
    			Pkt* pkt=&(pkts[i+j*32]);
    			int l=nf_id;
    			nfs.nf[l]->nf_logic(pkt,fs[i].fs[l]);
    		}
    		//fs[i].ptr=NULL;

    		//j+=packet_num;


    	}
    }
    Release_nfs(&nfs);

}



void gpu_nf_process(Pkt* h_pkts,Fs* h_fs,uint64_t service_chain,int packet_num,int *h_flow_size,struct d_rule* h_firewall_rules){

    Pkt* pkts;
    Fs* fs;
    int* flow_size;
   // int* flow_pos;
    struct d_rule* firewall_rules;

    hipHostGetDevicePointer((void **)&pkts, (void *)h_pkts, 0);
   // hipSetDeviceFlags(hipDeviceMapHost);
	hipHostGetDevicePointer((void **)&fs, (void *)h_fs, 0);
	// hipSetDeviceFlags(hipDeviceMapHost);
	hipHostGetDevicePointer((void **)&flow_size, (void *)h_flow_size, 0);
	// hipSetDeviceFlags(hipDeviceMapHost);
	hipHostGetDevicePointer((void **)&firewall_rules, (void *)h_firewall_rules, 0);
	//hipHostGetDevicePointer((void **)&flow_pos, (void *)h_flow_pos, 0);


	int threadsPerBlock = 256;
    int blocksPerGrid =(packet_num + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    hipDeviceSynchronize();

    Runtask<<<blocksPerGrid, threadsPerBlock>>>(pkts, fs, service_chain, packet_num,flow_size,firewall_rules);
    //hipDeviceSynchronize();


}


