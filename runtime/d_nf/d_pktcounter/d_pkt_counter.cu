#include "d_pkt_counter.ch"

#include "../d_base/d_network_function_register.h"


#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

bool d_registered_pkt_counter =
    d_static_nf_register::get_register().register_nf<d_pkt_counter, d_pkt_counter_fs>("pkt_counter", 1);
